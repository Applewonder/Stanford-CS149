#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256
#define MAX_BLOCKS 32768

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void
upsweep_kernel(int instance, int N, int next_pow2, int two_d, int* tmp, int* result) {
    int index = instance * MAX_BLOCKS + blockIdx.x * blockDim.x + threadIdx.x;
    int i = index * two_d * 2;
    // if (i < N && result[i+two_d-1] > 0) {
    //     int tmp = result[i+two_d*2-1] + result[i+two_d-1];
    //     if (tmp > 0) {
    //         result[i+two_d*2-1] = tmp;
    //     }
    // }
    if (i < next_pow2) {
        int index1 = i+two_d-1;
        int index2 = i+two_d*2-1;
        int* ptr1 = index1 < N ? result + index1 : tmp + index1 - N;
        int* ptr2 = index2 < N ? result + index2 : tmp + index2 - N;
        *ptr2 += *ptr1;
    }
}

__global__ void
downsweep_kernel(int instance, int N, int next_pow2, int two_d, int* tmp, int* result) {
    int index = instance * MAX_BLOCKS + blockIdx.x * blockDim.x + threadIdx.x;
    int i = index * two_d * 2;
    if (i < next_pow2) {
        int index1 = i+two_d-1;
        int index2 = i+two_d*2-1;
        int* ptr1 = index1 < N ? result + index1 : tmp + index1 - N;
        int* ptr2 = index2 < N ? result + index2 : tmp + index2 - N;
        int t = *ptr1;
        *ptr1 = *ptr2;
        *ptr2 += t;
    }
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    int* tmp = nullptr;
    int next_pow2 = nextPow2(N);
    int tmp_size = next_pow2 - N;
    int sign = hipMalloc(&tmp, tmp_size * sizeof(int));
    if (sign == hipErrorOutOfMemory) {
        fprintf(stderr, "Memory allocation failed\n");
        return;
    }

    // int N = end - start;
    // hipMemcpy(tmp, input, N*sizeof(int), hipMemcpyDeviceToDevice);
    // hipMemset(tmp+N, 0, (next_pow2-N)*sizeof(int));
    // return;
    // memmove(result, input, N*sizeof(int));
    hipMemcpy(result, input, N*sizeof(int), hipMemcpyDeviceToDevice);
    hipMemset(tmp, 0, tmp_size * sizeof(int));

    // upsweep phase
    for (int two_d = 1; two_d <= next_pow2/2; two_d*=2) {
        int num_blocks = (((next_pow2 + 2 * two_d - 1)/(2 * two_d)) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        // int two_dplus1 = 2*two_d;
        // parallel_for (int i = 0; i < N; i += two_dplus1) {
        //     output[i+two_dplus1-1] += output[i+two_d-1];
        // }
        for (int i = 0; i < num_blocks; i += MAX_BLOCKS) {
            int remaining = num_blocks - i;
            upsweep_kernel<<<remaining < MAX_BLOCKS ? remaining : MAX_BLOCKS, THREADS_PER_BLOCK>>>(i, N, next_pow2, two_d, tmp, result);
            hipDeviceSynchronize();
        }
    }
    // return;

    // result[N-1] = 0;
    // int zero = 0;
    // hipMemcpy(result+N-1, &zero, sizeof(int), hipMemcpyHostToDevice);
    // hipMemset(tmp+next_pow2-1, 0, sizeof(int));
    if (N < next_pow2) {
        hipMemset(tmp + next_pow2 - N - 1, 0, sizeof(int));
    } else {
        hipMemset(result + next_pow2 - 1, 0, sizeof(int));
    }

    // downsweep phase
    for (int two_d = next_pow2/2; two_d >= 1; two_d /= 2) {
        int num_blocks = (((next_pow2 + 2 * two_d - 1)/(2 * two_d)) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        // int two_dplus1 = 2*two_d;
        // parallel_for (int i = 0; i < N; i += two_dplus1) {
        //     int t = output[i+two_d-1];
        //     output[i+two_d-1] = output[i+two_dplus1-1];
        //     output[i+two_dplus1-1] += t;
        // }
        // downsweep_kernel<<<(((next_pow2+2*two_d-1)/(2*two_d)) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(next_pow2, two_d, tmp);
        for (int i = 0; i < num_blocks; i += MAX_BLOCKS) {
            int remaining = num_blocks - i;
            downsweep_kernel<<<remaining < MAX_BLOCKS ? remaining : MAX_BLOCKS, THREADS_PER_BLOCK>>>(i, N, next_pow2, two_d, tmp, result);
            hipDeviceSynchronize();
        }
    }

    // hipMemcpy(result, tmp, N*sizeof(int), hipMemcpyDeviceToDevice);
    hipFree(tmp);
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

__global__ void
find_repeats_kernel(int N, int* input, int* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        result[index] = (input[index] == input[index+1]) ? 1 : 0;
    }
}

__global__ void
find_repeats_index_kernel(int N, int* repeat_i, int* index, int* result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        if (repeat_i[i]) {
            result[index[i]] = i;
        }
    }
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    // parallel_for (int i = 0; i < length - 1; i++) {
    //     tmp_output[i] = (device_input[i] == device_input[i+1]) ? 1 : 0;
    // }
    int* tmp_output = nullptr;
    int* index = nullptr;
    hipMalloc(&tmp_output, length * sizeof(int));
    hipMalloc(&index, length * sizeof(int));

    find_repeats_kernel<<<(length + THREADS_PER_BLOCK - 2) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(length - 1, device_input, tmp_output);
    hipDeviceSynchronize();

    exclusive_scan(tmp_output, length, index);
    hipDeviceSynchronize();

    find_repeats_index_kernel<<<(length + THREADS_PER_BLOCK - 2) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(length - 1, tmp_output, index, device_output);
    hipDeviceSynchronize();
    
    int total_repeat_num = 0;
    hipMemcpy(&total_repeat_num, index + length - 1, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(tmp_output);
    hipFree(index);

    return total_repeat_num; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
